#include "hip/hip_runtime.h"
﻿

#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include "gdal.h"
#include "gdal_priv.h"
#include "gdalwarper.h"
#include <stdio.h>
#include<iostream>
#include "fusion.h"
#define num_thread 256
#define num_block 128
//__device__  void MultiplymBynandnByp(float *Mult1, float *Mult2, float *output, int m, int n, int p,int Idx,int Win_size1)
//{
//    int i,j,k;
//	for(i=0;i<m;i++)
//		for(j=0;j<p;j++)
//		{
//			output[i*p+j]=0;
//			for(k=0;k<n;k++)
//				output[i*p+j]+=Mult1[i*n+k+Idx*2*Win_size1*Win_size1]*Mult2[j+k*p+Idx*2*Win_size1*Win_size1];
//		}
//}
//__device__  void MatrixTransposemByn(float *input ,int m, int n, float *output,int Idx,int Win_size1)
//{
//    for(int i=0;i<n;i++)
//	for(int j=0;j<m;j++)
//	{
//		output[i*m+j+Idx*2*Win_size1*Win_size1] = input[j*n+i+Idx*2*Win_size1*Win_size1];
//	}
//}
//__device__ void CalcuRela(int num, float &a1,float &b1 ,float gamma,float *AA,float *BB,float *temp1,int Idx,int Win_size1,float B[2],float B1[2],float temp2[4],float temp3[2],float temp11[2],float temp22[4],float temp33[4],float tempB1[2],float tempB11[2], float output[4])
// {
//	 
//	 int i,j,k;
//	 a1=0,b1=0;
//	 for( i=0;i<2;i++)
//	 {
//		 for( j=0;j<num;j++)
//		 {
//			 temp1[i*num+j+Idx*2*Win_size1*Win_size1] = AA[j*2+i+Idx*2*Win_size1*Win_size1];
//		 }
//	 }
//	 for(i=0;i<2;i++)
//	 {
//		 for(j=0;j<2;j++)
//		 {
//			 temp2[i*2+j]=0;
//			 for(k=0;k<num;k++)
//				 temp2[i*2+j]+=temp1[i*num+k+Idx*2*Win_size1*Win_size1]*AA[j+k*2+Idx*2*Win_size1*Win_size1];
//		 }
//	 }
//	 // MatrixTransposemByn(AA,num,2,temp1,Idx,Win_size1);//(x,1)'
//	 // MultiplymBynandnByp(temp1,AA,temp2,2,num,2,Idx,Win_size1);//(x,1)'(x,1)
//	 for(i=0;i<2;i++)
//	 {
//		 for( j=0;j<1;j++)
//		 {
//			 temp11[i*1+j] = B1[j*2+i];
//		 }
//	 }
//	 //	 MultiplymBynandnByp(temp11,B,temp22,2,1,2,Idx,Win_size1);
//	 for(i=0;i<2;i++)
//	 {
//		 for(j=0;j<2;j++)
//		 {
//			 temp22[i*2+j]=0;
//			 for(k=0;k<1;k++)
//				 temp22[i*2+j]+=temp11[i*1+k]*B[j+k*2];
//		 }
//	 }
//	 for(i=0;i<2;i++)
//	 {
//		 for(j=0;j<2;j++)
//		 {
//			 temp33[i*2+j]=temp2[i*2+j]+temp22[i*2+j];
//		 }
//	 }
//
//	 for( i=0;i<4;i++)
//	 {
//		 output[i]=temp33[i];
//	 }
//	 output[2]=temp33[0]*temp33[3]-temp33[1]*temp33[2];
//	 output[0]=temp33[3]/output[2];
//	 output[3]=temp33[0]/output[2];
//	 output[1]=-temp33[2]/output[2];
//	 output[2]=-temp33[1]/output[2];
//	 for( i=0;i<4;i++)
//	 {
//		 temp33[i]=output[i];
//	 }
//	 // MultiplymBynandnByp(temp1,input2,temp3,2,num,1);
//	 for(i=0;i<2;i++)
//	 {
//		 for(j=0;j<1;j++)
//		 {
//			 temp3[i*1+j]=0;
//			 for(k=0;k<num;k++)
//				 temp3[i*1+j]+=temp1[i*num+k+Idx*2*Win_size1*Win_size1]*BB[j+k*1+Idx*Win_size1*Win_size1];
//		 }
//	 }
//	 //MatrixsumByp(temp3,temp11,tempB11,2,1);
//	 for(i=0;i<1;i++)
//	 {
//		 for(j=0;j<2;j++)
//		 {
//			 tempB11[i*2+j]=temp3[i*2+j]+temp11[i*2+j];
//		 }
//	 }
//	 //	 MultiplymBynandnByp(temp33,tempB11,Relationship,2,2,1);
//	 for(i=0;i<2;i++)
//	 {
//		 for(j=0;j<1;j++)
//		 {
//			 output[i+j]=0;
//			 for(k=0;k<2;k++)
//				 output[i+j]+=temp33[i*2+k]*tempB11[j+k];
//		 }
//	 }
//	 a1=output[0];
//	 b1=output[1];
// }
//__device__ void Location_pp(int *Location_P,float **BufferIn11,float **BufferIn22,float **BufferIn55,int i1,int j1,int rmin,int rmax,int smin,int smax,int nExWidth,int Height ,int Win_size1,int b,int BandNum,int &n)
//{
//	double threshold_d[10];
//	for (int i=0;i<BandNum;i++)
//	{
//		threshold_d[i]=0.01*pow(2,BufferIn11[i][ i1+nExWidth*j1]);
//	}
//
//	for(int r1=rmin;r1<=rmax;r1++)
//	{
//		for(int s1=smin;s1<=smax;s1++)
//		{  
//
//			int Result1=0;	
//			for(int i=0;i<BandNum;i++)
//			{  
//				if(fabs(BufferIn11[i][ r1+nExWidth*s1]-BufferIn11[i][ i1+nExWidth*j1])<=threshold_d[i])//ɸѡ
//				{
//					Result1++;
//				}
//				else
//					break;
//			}	
//
//			if(Result1==BandNum )
//			{	
//
//				double T1=fabs(BufferIn55[b][r1+nExWidth*s1]-BufferIn22[b][r1+nExWidth*s1]);
//				double S1=fabs(BufferIn11[b][r1+nExWidth*s1]-BufferIn22[b][r1+nExWidth*s1]);
//				if(S1<fabs(BufferIn11[b][i1+nExWidth*j1]-BufferIn22[b][i1+nExWidth*j1])+0.005||fabs(T1-fabs(BufferIn55[b][i1+nExWidth*j1]-BufferIn22[b][i1+nExWidth*j1]))<0.005)
//
//				{
//					Location_P[n+(b*nExWidth*Height+i1+nExWidth*j1)*100]=r1+nExWidth*s1;
//					n++;
//				}
//			}
//		}
//	}
//}
//__global__ void blending(float **BufferIn11,float **BufferIn22,float **BufferIn33,float **BufferIn44,float **BufferIn55, float **BufferOut,int Height,int Width, int Win_size1,int flag,double L_err,double M_err,double h1,int BandNum,double gamma,int *location_p)
//{
//	const int tid=threadIdx.x;
//	const int bid=blockIdx.x;
//	const int Idx=num_thread*bid+tid;
//	int Result1=0,m=0;
//	int b=Idx/(Height*Width);
//	int j=(Idx-(Height*Width)*b)/Width;
//	int i=(Idx-(Height*Width)*b)%Width;
//	int rmin,rmax,smin,smax;
//	int n1=0;
//	float T1=0,S1=0;
//	if(b<BandNum)
//	{
//		if(i-Win_size1/2<=0)
//			rmin=0;
//		else
//			rmin = i-Win_size1/2;
//
//		if(i+Win_size1/2>=Width-1)
//			rmax = Width-1;
//		else
//			rmax = i+Win_size1/2;
//
//		if(j-Win_size1/2<=0)
//			smin=0;
//		else
//			smin = j-Win_size1/2;
//
//		if(j+Win_size1/2>=Height-1)
//			smax = Height-1;
//		else
//			smax = j+Win_size1/2;
//		//Location_pp(location_p,BufferIn11,BufferIn22,BufferIn55,i, j, rmin, rmax, smin, smax, Width,Height,Win_size1, b, BandNum,n1);
//		float threshold_d[10];
//		int r1=rmin,s1=smin;
//		for ( m=0;m<BandNum;m++)
//		{
//			threshold_d[m]=0.01*pow((float)2.0,BufferIn11[m][ i+Width*j]);
//		}
//
//		for(r1=rmin;r1<=rmax;r1++)
//		{
//			for(s1=smin;s1<=smax;s1++)
//			{  
//
//				Result1=0;	
//				for( m=0;m<BandNum;m++)
//				{  
//					if(fabs(BufferIn11[m][ r1+Width*s1]-BufferIn11[m][ i+Width*j])<=threshold_d[m])//ɸѡ
//					{
//						Result1++;
//					}
//					else
//						break;
//				}	
//
//				if(Result1==BandNum )
//				{	
//
//					T1=fabs(BufferIn55[b][r1+Width*s1]-BufferIn22[b][r1+Width*s1]);
//					S1=fabs(BufferIn11[b][r1+Width*s1]-BufferIn22[b][r1+Width*s1]);
//					if(S1<fabs(BufferIn11[b][i+Width*j]-BufferIn22[b][i+Width*j])+0.005||fabs(T1-fabs(BufferIn55[b][i+Width*j]-BufferIn22[b][i+Width*j]))<0.005)
//
//					{
//						location_p[n1+(b*Width*Height+i+Width*j)*100]=r1+Width*s1;
//						n1++;
//					}
//				}
//			}
//		}
//		float weight1=0;
//		float Lst1=0;
//		float Total_w1=0;   
//		float Average1=0;
//		int k_h=0;
//		float S=0;
//		for( k_h=0;k_h<n1;k_h++)
//		{
//
//			Lst1=BufferIn11[b][location_p[k_h+(b*Width*Height+i+Width*j)*100]]+BufferIn33[b][location_p[k_h+(b*Width*Height+i+Width*j)*100]]-BufferIn22[b][location_p[k_h+(b*Width*Height+i+Width*j)*100]];
//			S=fabs(BufferIn22[b][location_p[k_h+(b*Width*Height+i+Width*j)*100]]-BufferIn33[b][i+Width*j]);
//			weight1=exp(-(S)/h1);
//			if(Lst1>0)
//			{
//				Average1+=weight1*Lst1;
//				Total_w1+=weight1;
//			}
//			else
//			{
//				Average1+=0;
//				Total_w1+=0;
//			}
//
//		}
//		BufferOut[b][j*Width+i]=Average1/Total_w1;
//		//BufferOut[b][j*Width+i]=0;
//		if(BufferOut[b][j*Width+i]<=0)
//		{
//			BufferOut[b][j*Width+i]=BufferIn33[b][j*Width+i];
//		}
//	}
//}
//__global__ void blending1(float **BufferIn11,float **BufferIn22,float **BufferIn33,float **BufferIn44,float **BufferIn55, float **BufferOut,int Height,int Width, int Win_size1,int flag,double L_err,double M_err,double h1,int BandNum,double gamma,int *location_p,float *Changed_BufferIn11,float *Changed_BufferIn33)
//{
//	const int tid=threadIdx.x;
//	const int bid=blockIdx.x;
//	const int Idx=num_thread*bid+tid;
//	int Result1=0,m=0;
//	int b=Idx/(Height*Width);
//	int j=(Idx-(Height*Width)*b)/Width;
//	int i=(Idx-(Height*Width)*b)%Width;
//	int rmin,rmax,smin,smax;
//	int n1=0;
//	float T1=0,S1=0;
//	float threshold_d[10];
//	float weight1=0;
//	float Lst1=0;
//	float Total_w1=0;
//	float Total_w2=0;
//	float Average2=0;
//	float Average1=0;
//	int k_h=0;
//	float S=0;
//	int r1,s1;
//	float Lst2;
//	float weight2;
//	double Aver11;
//	double Aver22;
//	float T1_weight;
//	float T2_weight;
//	float revise1,revise2;
//	float revise_w1,revise_w2;
//	for(int kkk=Idx;kkk<Width*Height*BandNum;kkk=kkk+num_thread*num_block)
//	{
//		Result1=0,m=0;
//		b=kkk/(Height*Width);
//		j=(kkk-(Height*Width)*b)/Width;
//		i=(kkk-(Height*Width)*b)%Width;
//		n1=0;
//		revise1=0,revise2=0,revise_w1=0,revise_w2;
//		T1=0,S1=0;
//		weight1=0;
//		weight2=0;
//		Lst1=0;
//		Lst2=0;
//		Total_w1=0;  
//		Total_w2=0;
//		Average1=0;
//		Average2=0;
//		k_h=0;
//		S=0;
//		Aver11=0;
//		Aver22=0;
//		if(b<BandNum)
//		{
//			if(i-Win_size1/2<=0)
//				rmin=0;
//			else
//				rmin = i-Win_size1/2;
//
//			if(i+Win_size1/2>=Width-1)
//				rmax = Width-1;
//			else
//				rmax = i+Win_size1/2;
//
//			if(j-Win_size1/2<=0)
//				smin=0;
//			else
//				smin = j-Win_size1/2;
//
//			if(j+Win_size1/2>=Height-1)
//				smax = Height-1;
//			else
//				smax = j+Win_size1/2;
//			//Location_pp(location_p,BufferIn11,BufferIn22,BufferIn55,i, j, rmin, rmax, smin, smax, Width,Height,Win_size1, b, BandNum,n1);
//			r1=rmin,s1=smin;
//			for ( m=0;m<BandNum;m++)
//			{
//				threshold_d[m]=0.01*pow(2,BufferIn11[m][ i+Width*j]);
//			}
//
//			for(r1=rmin;r1<=rmax;r1++)
//			{
//				for(s1=smin;s1<=smax;s1++)
//				{  
//
//					Result1=0;	
//					for( m=0;m<BandNum;m++)
//					{  
//						if(fabs(BufferIn11[m][ r1+Width*s1]-BufferIn11[m][ i+Width*j])<=threshold_d[m])//ɸѡ
//						{
//							Result1++;
//						}
//						else
//							break;
//					}	
//
//					if(Result1==BandNum )
//					{	
//
//						T1=fabs(BufferIn55[b][r1+Width*s1]-BufferIn22[b][r1+Width*s1]);
//						S1=fabs(BufferIn11[b][r1+Width*s1]-BufferIn22[b][r1+Width*s1]);
//						if(S1<fabs(BufferIn11[b][i+Width*j]-BufferIn22[b][i+Width*j])+0.005||fabs(T1-fabs(BufferIn55[b][i+Width*j]-BufferIn22[b][i+Width*j]))<0.005)
//
//						{
//							location_p[n1+Idx*Win_size1*Win_size1]=r1+Width*s1;
//							n1++;
//						}
//					}
//				}
//			}
//			for( k_h=0;k_h<n1;k_h++)
//			{
//
//				Lst1=BufferIn11[b][location_p[k_h+Idx*Win_size1*Win_size1]]+BufferIn55[b][location_p[k_h+Idx*Win_size1*Win_size1]]-BufferIn22[b][location_p[k_h+Idx*Win_size1*Win_size1]];
//				S=fabs(BufferIn22[b][location_p[k_h+Idx*Win_size1*Win_size1]]-BufferIn55[b][i+Width*j]);
//				weight1=exp(-(S)/h1);
//				revise1+= weight1*BufferIn11[b][location_p[k_h+Idx*Win_size1*Win_size1]];
//				revise_w1+= weight1;
//				if(Lst1>0)
//				{
//					Average1+=weight1*Lst1;
//					Total_w1+=weight1;
//				}
//				else
//				{
//					Average1+=0;
//					Total_w1+=0;
//				}
//
//			}
//
//
//
//
//
//
//			for ( m=0;m<BandNum;m++)
//			{
//				threshold_d[m]=0.01*pow(2,BufferIn33[m][ i+Width*j]);
//			}
//			n1=0;
//			for(r1=rmin;r1<=rmax;r1++)
//			{
//				for(s1=smin;s1<=smax;s1++)
//				{  
//
//					Result1=0;	
//					for( m=0;m<BandNum;m++)
//					{  
//						if(fabs(BufferIn33[m][ r1+Width*s1]-BufferIn33[m][ i+Width*j])<=threshold_d[m])//ɸѡ
//						{
//							Result1++;
//						}
//						else
//							break;
//					}	
//
//					if(Result1==BandNum )
//					{	
//
//						T1=fabs(BufferIn55[b][r1+Width*s1]-BufferIn44[b][r1+Width*s1]);
//						S1=fabs(BufferIn33[b][r1+Width*s1]-BufferIn44[b][r1+Width*s1]);
//						if(S1<fabs(BufferIn33[b][i+Width*j]-BufferIn44[b][i+Width*j])+0.005||fabs(T1-fabs(BufferIn55[b][i+Width*j]-BufferIn44[b][i+Width*j]))<0.005)
//
//						{
//							location_p[n1+Idx*Win_size1*Win_size1]=r1+Width*s1;
//							n1++;
//						}
//					}
//				}
//			}
//			for( k_h=0;k_h<n1;k_h++)
//			{
//
//				Lst2=BufferIn33[b][location_p[k_h+Idx*Win_size1*Win_size1]]+BufferIn55[b][location_p[k_h+Idx*Win_size1*Win_size1]]-BufferIn44[b][location_p[k_h+Idx*Win_size1*Win_size1]];
//				S=fabs(BufferIn44[b][location_p[k_h+Idx*Win_size1*Win_size1]]-BufferIn55[b][i+Width*j]);
//				weight2=exp(-(S)/h1);
//				revise2+= weight2*BufferIn33[b][location_p[k_h+Idx*Win_size1*Win_size1]];
//				revise_w2+= weight2;
//				if(Lst2>0)
//				{
//					Average2+=weight2*Lst2;
//					Total_w2+=weight2;
//				}
//				else
//				{
//					Average2+=0;
//					Total_w2+=0;
//				}
//
//			}
//			for(int r1=rmin;r1<=rmax;r1++)
//			{
//				for(int s1=smin;s1<=smax;s1++)
//				{  
//					Aver11+=fabs(BufferIn22[b][r1+Width*s1]-BufferIn55[b][r1+Width*s1]);
//					Aver22+=fabs(BufferIn44[b][r1+Width*s1]-BufferIn55[b][r1+Width*s1]);	
//				}
//			}
//			T1_weight=1/Aver11/(1/Aver11+1/Aver22);
//			T2_weight=1/Aver22/(1/Aver11+1/Aver22);	
//			BufferOut[b][j*Width+i]=T1_weight*Average1/Total_w1+T2_weight*Average2/Total_w2;
//			//BufferOut[b][j*Width+i]=0;
//			if(BufferOut[b][j*Width+i]<=0||BufferOut[b][j*Width+i]==NULL)
//			{
//				BufferOut[b][j*Width+i]=T1_weight*revise1/revise_w1+T2_weight*revise2/revise_w2;
//			}
//		}
//	}
//}
__global__ void blending2(float **BufferIn11,float **BufferIn22,float **BufferIn33,float **BufferIn44,float **BufferIn55, float **BufferOut,int Height,int Width, int Win_size1,int flag,double L_err,double M_err,double h1,int BandNum,double gamma,int *location_p,float *Changed_BufferIn11,float *Changed_BufferIn33,float *GausKernel,int p_Para,float d_Para,int pacthSize )
{
	const int tid=threadIdx.x;
	const int bid=blockIdx.x;
	const int Idx=num_thread*bid+tid;
	int Result1=0,m=0;
	int b=Idx/(Height*Width);
	int j=(Idx-(Height*Width)*b)/Width;
	int i=(Idx-(Height*Width)*b)%Width;
	int rmin,rmax,smin,smax;
	int n1=0;
	float T1=0,S1=0;
	float threshold_d[10];
	float weight1=0;
	float Lst1=0;
	float Total_w1=0;
	float Total_w2=0;
	float Average2=0;
	float Average1=0;
	int k_h=0;
	float S=0;
	int r1,s1;
	float Lst2;
	float weight2;
	double Aver11;
	double Aver22;
	float T1_weight;
	float T2_weight;
	float revise1,revise2;
	float revise_w1,revise_w2;
	float value;
	float sumGS;
	for(int kkk=Idx;kkk<Width*Height*BandNum;kkk=kkk+num_thread*num_block)
	{
		Result1=0,m=0;
		b=kkk/(Height*Width);
		j=(kkk-(Height*Width)*b)/Width;
		i=(kkk-(Height*Width)*b)%Width;
		n1=0;
		revise1=0;
		revise2=0;
		revise_w1=0;
		revise_w2=0;
		T1=0;
		S1=0;
		weight1=0;
		weight2=0;
		Lst1=0;
		Lst2=0;
		Total_w1=0;  
		Total_w2=0;
		Average1=0;
		Average2=0;
		k_h=0;
		S=0;
		Aver11=0;
		Aver22=0;
		if(b<BandNum)
		{
			if(fabs(BufferIn55[b][j*Width+i])<1e-6)
			{
				BufferOut[b][j*Width+i]=0;
			}
			else
			{
			if(i-Win_size1/2<=0)
				rmin=0;
			else
				rmin = i-Win_size1/2;

			if(i+Win_size1/2>=Width-1)
				rmax = Width-1;
			else
				rmax = i+Win_size1/2;

			if(j-Win_size1/2<=0)
				smin=0;
			else
				smin = j-Win_size1/2;

			if(j+Win_size1/2>=Height-1)
				smax = Height-1;
			else
				smax = j+Win_size1/2;
			//Location_pp(location_p,BufferIn11,BufferIn22,BufferIn55,i, j, rmin, rmax, smin, smax, Width,Height,Win_size1, b, BandNum,n1);
			r1=rmin,s1=smin;
			for ( m=0;m<BandNum;m++)
			{
				if (p_Para == 1||p_Para == 2)       //prodType == 1表示为反射率产品,prodType == 2表示为指数类产品
		       {
				threshold_d[m]=d_Para*pow((float)2,BufferIn11[m][ i+Width*j]);
				}
				else if(p_Para == 3)
				{
					threshold_d[m]=d_Para*pow((float)2,BufferIn11[m][ i+Width*j]/1000);
				}
			}
			float thresSpecHomo = sqrt(L_err*L_err+M_err*M_err);
	               float thresTempDiff = sqrt(2.0)*M_err;
			for(r1=rmin;r1<=rmax;r1++)
			{
				for(s1=smin;s1<=smax;s1++)
				{  

					Result1=0;	
					for( m=0;m<BandNum;m++)
					{  
						if(fabs(BufferIn11[m][ r1+Width*s1]-BufferIn11[m][ i+Width*j])<=threshold_d[m])//ɸѡ
						{
							Result1++;
						}
						else
							break;
					}	

					if(Result1==BandNum )
					{	

						T1=fabs(BufferIn55[b][r1+Width*s1]-BufferIn22[b][r1+Width*s1]);
						S1=fabs(BufferIn11[b][r1+Width*s1]-BufferIn22[b][r1+Width*s1]);
						if(S1<fabs(BufferIn11[b][i+Width*j]-BufferIn22[b][i+Width*j])+thresSpecHomo||fabs(T1-fabs(BufferIn55[b][i+Width*j]-BufferIn22[b][i+Width*j]))<thresTempDiff)

						{
							location_p[n1+Idx*Win_size1*Win_size1]=r1+Width*s1;
							n1++;
						}
					}
				}
			}
			sumGS=0;
			if(pacthSize!=1)
			{
			for(m=0;m<=pacthSize/2;m++)
			{
				value=1/((2.0*(double)m+1)*(2*(double)m+1));
				for (int r=-m; r<=m; r++)
				{
					for (int c=-m; c<=m; c++)
					{
						GausKernel[Idx*pacthSize*pacthSize+(pacthSize/2+r)*pacthSize+(pacthSize/2+c)] += value;			
					}
				}
			}
			for (m=0; m<pacthSize*pacthSize; m++)
			{
				GausKernel[Idx*pacthSize*pacthSize+m] = GausKernel[Idx*pacthSize*pacthSize+m]/(pacthSize/2);
				sumGS+= GausKernel[Idx*pacthSize*pacthSize+m];
			}

			for (m=0; m<pacthSize*pacthSize; m++)
			{
				GausKernel[Idx*pacthSize*pacthSize+m] = GausKernel[Idx*pacthSize*pacthSize+m]/sumGS;
			}
			}
			for( k_h=0;k_h<n1;k_h++)
			{

				Lst1=Changed_BufferIn11[b*Height*Width+location_p[k_h+Idx*Win_size1*Win_size1]];
				if((p_Para==1&&Lst1>0&&Lst1<=1)||(p_Para==2&&Lst1>=-1&&Lst1<=1)||(p_Para==3&&Lst1!=0))
				{
				if(pacthSize==1)
				{
				S=fabs(BufferIn22[b][location_p[k_h+Idx*Win_size1*Win_size1]]-BufferIn55[b][i+Width*j]);
				}
				else
				{
					int tarPRow, tarPCol;
					int simPRow, simPCol;
					int tarPLoc, simPLoc, kerPLoc;
					float sumCa= 0;
					int row=location_p[k_h+Idx*Win_size1*Win_size1]/Width;
					int colume=location_p[k_h+Idx*Win_size1*Win_size1]%Width;
					for (int rowOffset=-pacthSize/2; rowOffset<=pacthSize/2; rowOffset++)
					{
						for (int colOffset=-pacthSize/2; colOffset<=pacthSize/2; colOffset++)
						{
							if (j+rowOffset < 0)
								tarPRow = -(j+rowOffset)-1;
							else if (j+rowOffset > Height-1)
								tarPRow = 2*Height-1-(j+rowOffset);
							else
								tarPRow = j+rowOffset;

							if (i+colOffset < 0)
								tarPCol = -(i+colOffset)-1;
							else if (i+colOffset > Width-1)
								tarPCol = 2*Width-1-(i+colOffset);
							else
								tarPCol = i+colOffset;


							/* 对相似像元图块中的像元的行列范围进行判断，对超出图幅行列范围的像元进行填补 */
							if (row+rowOffset < 0)
								simPRow = -(row+rowOffset)-1;
							else if (row+rowOffset > Height-1)
								simPRow = 2*Height-1-(row+rowOffset);
							else
								simPRow = row+rowOffset;

							if (colume+colOffset < 0)
								simPCol = -(colume+colOffset)-1;
							else if (colume+colOffset > Width-1)
								simPCol = 2*Width-1-(colume+colOffset);
							else
								simPCol = colume+colOffset;

							/* 计算块的相似度 */
							tarPLoc = tarPRow * Width + tarPCol;
							simPLoc = simPRow * Width + simPCol;
							kerPLoc = (pacthSize/2+rowOffset) *pacthSize +(pacthSize/2+colOffset);
							sumCa+= GausKernel[kerPLoc+Idx*pacthSize*pacthSize]*fabs(BufferIn22[b][simPLoc]-BufferIn55[b][tarPLoc]);
						}
					}
					S=sumCa;

				}
				weight1=exp(-(S)/(h1*h1));
				revise1+= weight1*BufferIn11[b][location_p[k_h+Idx*Win_size1*Win_size1]];
				revise_w1+= weight1;
					Average1+=weight1*Lst1;
					Total_w1+=weight1;

				}

			}






			for ( m=0;m<BandNum;m++)
			{
				if (p_Para == 1||p_Para == 2)       //prodType == 1表示为反射率产品,prodType == 2表示为指数类产品
		       {
				threshold_d[m]=d_Para*pow((float)2,BufferIn33[m][ i+Width*j]);
				}
				else if(p_Para == 3)
				{
					threshold_d[m]=d_Para*pow((float)2,BufferIn33[m][ i+Width*j]/1000);
				}
			}
			n1=0;
			for(r1=rmin;r1<=rmax;r1++)
			{
				for(s1=smin;s1<=smax;s1++)
				{  

					Result1=0;	
					for( m=0;m<BandNum;m++)
					{  
						if(fabs(BufferIn33[m][ r1+Width*s1]-BufferIn33[m][ i+Width*j])<=threshold_d[m])//ɸѡ
						{
							Result1++;
						}
						else
							break;
					}	
				
					if(Result1==BandNum )
					{	

						T1=fabs(BufferIn55[b][r1+Width*s1]-BufferIn44[b][r1+Width*s1]);
						S1=fabs(BufferIn33[b][r1+Width*s1]-BufferIn44[b][r1+Width*s1]);
						if(S1<fabs(BufferIn33[b][i+Width*j]-BufferIn44[b][i+Width*j])+thresSpecHomo||fabs(T1-fabs(BufferIn55[b][i+Width*j]-BufferIn44[b][i+Width*j]))<thresTempDiff)

						{
							location_p[n1+Idx*Win_size1*Win_size1]=r1+Width*s1;
							n1++;
						}
					}
				}
			}
			for( k_h=0;k_h<n1;k_h++)
			{

				Lst2=Changed_BufferIn33[b*Height*Width+location_p[k_h+Idx*Win_size1*Win_size1]];
				if((p_Para==1&&Lst2>0&&Lst2<=1)||(p_Para==2&&Lst2>=-1&&Lst2<=1)||(p_Para==3&&Lst2!=0))
				{
				if(pacthSize==1)
				{
				S=fabs(BufferIn44[b][location_p[k_h+Idx*Win_size1*Win_size1]]-BufferIn55[b][i+Width*j]);
				}
				else
				{
					int tarPRow, tarPCol;
					int simPRow, simPCol;
					int tarPLoc, simPLoc, kerPLoc;
					float sumCa= 0;
					int row=location_p[k_h+Idx*Win_size1*Win_size1]/Width;
					int colume=location_p[k_h+Idx*Win_size1*Win_size1]%Width;
					for (int rowOffset=-pacthSize/2; rowOffset<=pacthSize/2; rowOffset++)
					{
						for (int colOffset=-pacthSize/2; colOffset<=pacthSize/2; colOffset++)
						{
							if (j+rowOffset < 0)
								tarPRow = -(j+rowOffset)-1;
							else if (j+rowOffset > Height-1)
								tarPRow = 2*Height-1-(j+rowOffset);
							else
								tarPRow = j+rowOffset;

							if (i+colOffset < 0)
								tarPCol = -(i+colOffset)-1;
							else if (i+colOffset > Width-1)
								tarPCol = 2*Width-1-(i+colOffset);
							else
								tarPCol = i+colOffset;


							/* 对相似像元图块中的像元的行列范围进行判断，对超出图幅行列范围的像元进行填补 */
							if (row+rowOffset < 0)
								simPRow = -(row+rowOffset)-1;
							else if (row+rowOffset > Height-1)
								simPRow = 2*Height-1-(row+rowOffset);
							else
								simPRow = row+rowOffset;

							if (colume+colOffset < 0)
								simPCol = -(colume+colOffset)-1;
							else if (colume+colOffset > Width-1)
								simPCol = 2*Width-1-(colume+colOffset);
							else
								simPCol = colume+colOffset;

							/* 计算块的相似度 */
							tarPLoc = tarPRow * Width + tarPCol;
							simPLoc = simPRow * Width + simPCol;
							kerPLoc = (pacthSize/2+rowOffset) *pacthSize +(pacthSize/2+colOffset);
							sumCa+= GausKernel[kerPLoc+Idx*pacthSize*pacthSize]*fabs(BufferIn44[b][simPLoc]-BufferIn55[b][tarPLoc]);
						}
					}
					S=sumCa;

				}
				weight2=exp(-(S)/(h1*h1));
				revise2+= weight2*BufferIn33[b][location_p[k_h+Idx*Win_size1*Win_size1]];
				revise_w2+= weight2;
					Average2+=weight2*Lst2;
					Total_w2+=weight2;
				}

			}
			for(int r1=rmin;r1<=rmax;r1++)
			{
				for(int s1=smin;s1<=smax;s1++)
				{  
					Aver11+=fabs(BufferIn22[b][r1+Width*s1]-BufferIn55[b][r1+Width*s1]);
					Aver22+=fabs(BufferIn44[b][r1+Width*s1]-BufferIn55[b][r1+Width*s1]);	
				}
			}
			T1_weight=1/Aver11/(1/Aver11+1/Aver22);
			T2_weight=1/Aver22/(1/Aver11+1/Aver22);	
			BufferOut[b][j*Width+i]=T1_weight*Average1/Total_w1+T2_weight*Average2/Total_w2;
			//BufferOut[b][j*Width+i]=0;
			if(BufferOut[b][j*Width+i]<=0||BufferOut[b][j*Width+i]==NULL)
			{
				BufferOut[b][j*Width+i]=T1_weight*revise1/revise_w1+T2_weight*revise2/revise_w2;
			}
			}
		}
	}
}
__global__ void blending3(float **BufferIn11,float **BufferIn22,float **BufferIn55, float **BufferOut,int Height,int Width, int Win_size1,int flag,double L_err,double M_err,double h1,int BandNum,double gamma,int *location_p,float *Changed_BufferIn11,float *GausKernel,int p_Para,float d_Para,int pacthSize )
{
	const int tid=threadIdx.x;
	const int bid=blockIdx.x;
	const int Idx=num_thread*bid+tid;
	int Result1=0,m=0;
	int b=Idx/(Height*Width);
	int j=(Idx-(Height*Width)*b)/Width;
	int i=(Idx-(Height*Width)*b)%Width;
	int rmin,rmax,smin,smax;
	int n1=0;
	float T1=0,S1=0;
	float threshold_d[10];
	float weight1=0;
	float Lst1=0;
	float Total_w1=0;
	float Average1=0;
	int k_h=0;
	float S=0;
	int r1,s1;
	float revise1;
	float revise_w1;
	 float sumGS;
	 float value;
	for(int kkk=Idx;kkk<Width*Height*BandNum;kkk=kkk+num_thread*num_block)
	{
		value=0;
		 sumGS=0;
		Result1=0,m=0;
		b=kkk/(Height*Width);
		j=(kkk-(Height*Width)*b)/Width;
		i=(kkk-(Height*Width)*b)%Width;
		n1=0;
		revise1=0;
		revise_w1=0;
		T1=0;
		S1=0;
		weight1=0;
		Lst1=0;
		Total_w1=0;  
		Average1=0;
		k_h=0;
		S=0;
		if(b<BandNum)
		{
			if(fabs(BufferIn55[b][j*Width+i])<1e-6)
			{
				BufferOut[b][j*Width+i]=0;
			}
			else
			{
			if(i-Win_size1/2<=0)
				rmin=0;
			else
				rmin = i-Win_size1/2;

			if(i+Win_size1/2>=Width-1)
				rmax = Width-1;
			else
				rmax = i+Win_size1/2;

			if(j-Win_size1/2<=0)
				smin=0;
			else
				smin = j-Win_size1/2;

			if(j+Win_size1/2>=Height-1)
				smax = Height-1;
			else
				smax = j+Win_size1/2;
			//Location_pp(location_p,BufferIn11,BufferIn22,BufferIn55,i, j, rmin, rmax, smin, smax, Width,Height,Win_size1, b, BandNum,n1);
			r1=rmin,s1=smin;
			for ( m=0;m<BandNum;m++)
			{
				if(p_Para==1||p_Para==2)
				{
				threshold_d[m]=d_Para*pow((float)2,BufferIn11[m][ i+Width*j]);
				}
				else if(p_Para==3)
				{
					threshold_d[m]=d_Para*pow((float)2,BufferIn11[m][ i+Width*j]/1000);
				}
			}

			for(r1=rmin;r1<=rmax;r1++)
			{
				for(s1=smin;s1<=smax;s1++)
				{  

					Result1=0;	
					for( m=0;m<BandNum;m++)
					{  
						if(fabs(BufferIn11[m][ r1+Width*s1]-BufferIn11[m][ i+Width*j])<=threshold_d[m])//ɸѡ
						{
							Result1++;
						}
						else
							break;
					}	

					if(Result1==BandNum )
					{	

						T1=fabs(BufferIn55[b][r1+Width*s1]-BufferIn22[b][r1+Width*s1]);
						S1=fabs(BufferIn11[b][r1+Width*s1]-BufferIn22[b][r1+Width*s1]);
						if(S1<fabs(BufferIn11[b][i+Width*j]-BufferIn22[b][i+Width*j])+sqrt(M_err*M_err+L_err*L_err)||fabs(T1-fabs(BufferIn55[b][i+Width*j]-BufferIn22[b][i+Width*j]))<M_err*sqrt(2.0))

						{
							location_p[n1+Idx*Win_size1*Win_size1]=r1+Width*s1;
							n1++;
						}
					}
				}
			}
			sumGS=0;
			if(pacthSize!=1)
			{
			for(m=0;m<=pacthSize/2;m++)
			{
				value=1.0/((2.0*(double)m+1)*(2*(double)m+1));
				for (int r=-m; r<=m; r++)
				{
					for (int c=-m; c<=m; c++)
					{
						GausKernel[Idx*pacthSize*pacthSize+(pacthSize/2+r)*pacthSize+(pacthSize/2+c)] += value;			
					}
				}
			}
			for (m=0; m<pacthSize*pacthSize; m++)
			{
				GausKernel[Idx*pacthSize*pacthSize+m] = GausKernel[Idx*pacthSize*pacthSize+m]/(pacthSize/2);
				sumGS+= GausKernel[Idx*pacthSize*pacthSize+m];
			}

			for (m=0; m<pacthSize*pacthSize; m++)
			{
				GausKernel[Idx*pacthSize*pacthSize+m] = GausKernel[Idx*pacthSize*pacthSize+m]/sumGS;
			}
			}
			for( k_h=0;k_h<n1;k_h++)
			{

				Lst1=Changed_BufferIn11[b*Height*Width+location_p[k_h+Idx*Win_size1*Win_size1]];
				if((p_Para==1&&Lst1>0&&Lst1<=1)||(p_Para==2&&Lst1>=-1&&Lst1<=1)||(p_Para==3&&Lst1!=0))
				{
				if(pacthSize==1)
				{
				S=fabs(BufferIn22[b][location_p[k_h+Idx*Win_size1*Win_size1]]-BufferIn55[b][i+Width*j]);
				}
				else
				{
					int tarPRow, tarPCol;
					int simPRow, simPCol;
					int tarPLoc, simPLoc, kerPLoc;
					float sumCa= 0;
					int row=location_p[k_h+Idx*Win_size1*Win_size1]/Width;
					int colume=location_p[k_h+Idx*Win_size1*Win_size1]% Width;
					for (int rowOffset=-pacthSize/2; rowOffset<=pacthSize/2; rowOffset++)
					{
						for (int colOffset=-pacthSize/2; colOffset<=pacthSize/2; colOffset++)
						{
							if (j+rowOffset < 0)
								tarPRow = -(j+rowOffset)-1;
							else if (j+rowOffset > Height-1)
								tarPRow = 2*Height-1-(j+rowOffset);
							else
								tarPRow = j+rowOffset;

							if (i+colOffset < 0)
								tarPCol = -(i+colOffset)-1;
							else if (i+colOffset > Width-1)
								tarPCol = 2*Width-1-(i+colOffset);
							else
								tarPCol = i+colOffset;


							/* 对相似像元图块中的像元的行列范围进行判断，对超出图幅行列范围的像元进行填补 */
							if (row+rowOffset < 0)
								simPRow = -(row+rowOffset)-1;
							else if (row+rowOffset > Height-1)
								simPRow = 2*Height-1-(row+rowOffset);
							else
								simPRow = row+rowOffset;

							if (colume+colOffset < 0)
								simPCol = -(colume+colOffset)-1;
							else if (colume+colOffset > Width-1)
								simPCol = 2*Width-1-(colume+colOffset);
							else
								simPCol = colume+colOffset;

							/* 计算块的相似度 */
							tarPLoc = tarPRow * Width + tarPCol;
							simPLoc = simPRow * Width + simPCol;
							kerPLoc = (pacthSize/2+rowOffset) *pacthSize +(pacthSize/2+colOffset);
							sumCa+= GausKernel[kerPLoc+Idx*pacthSize*pacthSize]*fabs(BufferIn22[b][simPLoc]-BufferIn55[b][tarPLoc]);
						}
					}
					S=sumCa;

				}
				weight1=exp(-(S)/(h1*h1));
				revise1+= weight1*BufferIn11[b][location_p[k_h+Idx*Win_size1*Win_size1]];
				revise_w1+= weight1;
					Average1+=weight1*Lst1;
					Total_w1+=weight1;
			}

			}



				
			BufferOut[b][j*Width+i]=Average1/Total_w1;
			//BufferOut[b][j*Width+i]=0;
			if(BufferOut[b][j*Width+i]<0||BufferOut[b][j*Width+i]==NULL)
			{
				BufferOut[b][j*Width+i]=revise1/revise_w1;
			}
		}
		}
	}
}
__global__ void limit_a_CalcuRela(float **BufferIn11,float **BufferIn22,float **BufferIn33,float **BufferIn44,float **BufferIn55, int Height,int Width, int Win_size1,int flag,double L_err,double M_err,double h1,int BandNum,double gamma,int *location_p,float *Changed_BufferIn11,float *Changed_BufferIn33,int p_Para,float d_Para)
{
	const int tid=threadIdx.x;
	const int bid=blockIdx.x;
	const int Idx=num_thread*bid+tid;
	int Result1=0,m=0;
	int b=Idx/(Height*Width);
	int j=(Idx-(Height*Width)*b)/Width;
	int i=(Idx-(Height*Width)*b)%Width;
	int rmin,rmax,smin,smax;
	int n1=0,n2=0;
	float T1=0,S1=0;
   float threshold_d[10];
	//float B1[2]={1,0};
	//float B2[2]={gamma,0};
	//float temp2[4],temp22[4],temp33[4],tempB1[2],tempB11[2];
	//float temp3[2],temp11[2];
	int k_h=0;
	//float output[4];
	int r1,s1;
		float sumx,sumy,sumxy,sumx2;
	float aa,bb,aa2,bb2;
	for(int kkk=Idx;kkk<Width*Height*BandNum;kkk=kkk+num_thread*num_block)
	{
	//	B1[0]=1,B1[1]=0;
	//	B2[0]=1,B2[1]=0;
		aa=0,bb=0;
		aa2=0,bb2=0;
		Result1=0,m=0;
		b=kkk/(Height*Width);
		j=(kkk-(Height*Width)*b)/Width;
		i=(kkk-(Height*Width)*b)%Width;
		n1=0;
		n2=0;
		T1=0,S1=0;
		k_h=0;
			sumx=0;
			sumy=0;
		     sumxy=0;
			sumx2=0;
		if(b<BandNum)
		{
			if(fabs(BufferIn11[b][j*Width+i])<1e-6&&fabs(BufferIn33[b][j*Width+i])<1e-6)
			{
				Changed_BufferIn11[kkk]=0;
			     Changed_BufferIn33[kkk]=0;
			}
			else
			{
			if(i-Win_size1/2<=0)
				rmin=0;
			else
				rmin = i-Win_size1/2;

			if(i+Win_size1/2>=Width-1)
				rmax = Width-1;
			else
				rmax = i+Win_size1/2;

			if(j-Win_size1/2<=0)
				smin=0;
			else
				smin = j-Win_size1/2;

			if(j+Win_size1/2>=Height-1)
				smax = Height-1;
			else
				smax = j+Win_size1/2;
			//Location_pp(location_p,BufferIn11,BufferIn22,BufferIn55,i, j, rmin, rmax, smin, smax, Width,Height,Win_size1, b, BandNum,n1);
			r1=rmin,s1=smin;
			for ( m=0;m<BandNum;m++)
			{
				if (p_Para == 1||p_Para == 2)       //prodType == 1表示为反射率产品,prodType == 2表示为指数类产品
		       {
				threshold_d[m]=d_Para*pow((float)2,BufferIn11[m][ i+Width*j]);
				}
				else if(p_Para==3)
				{
					threshold_d[m]=d_Para*pow((float)2,BufferIn11[m][ i+Width*j]/1000);
				}

			}

			for(r1=rmin;r1<=rmax;r1++)
			{
				for(s1=smin;s1<=smax;s1++)
				{  

					Result1=0;	
					for( m=0;m<BandNum;m++)
					{  
						if(fabs(BufferIn11[m][ r1+Width*s1]-BufferIn11[m][ i+Width*j])<=threshold_d[m])//ɸѡ
						{
							Result1++;
						}
						else
							break;
					}	

					if(Result1==BandNum )
					{	

						T1=fabs(BufferIn55[b][r1+Width*s1]-BufferIn22[b][r1+Width*s1]);
						S1=fabs(BufferIn11[b][r1+Width*s1]-BufferIn22[b][r1+Width*s1]);
						if(S1<fabs(BufferIn11[b][i+Width*j]-BufferIn22[b][i+Width*j])+sqrt(M_err*M_err+L_err*L_err)||fabs(T1-fabs(BufferIn55[b][i+Width*j]-BufferIn22[b][i+Width*j]))<M_err*sqrt(2.0))

						{
							location_p[n1+Idx*Win_size1*Win_size1]=r1+Width*s1;
							n1++;
						}
					}
				}
			}
			for ( m=0;m<BandNum;m++)
			{
				if (p_Para == 1||p_Para == 2)       //prodType == 1表示为反射率产品,prodType == 2表示为指数类产品
		       {
				threshold_d[m]=d_Para*pow((float)2,BufferIn33[m][ i+Width*j]);
				}
				else if(p_Para==3)
				{
					threshold_d[m]=d_Para*pow((float)2,BufferIn33[m][ i+Width*j]/1000);
				}
			}
			for(r1=rmin;r1<=rmax;r1++)
			{
				for(s1=smin;s1<=smax;s1++)
				{  

					Result1=0;	
					for( m=0;m<BandNum;m++)
					{  
						if(fabs(BufferIn33[m][ r1+Width*s1]-BufferIn33[m][ i+Width*j])<=threshold_d[m])//ɸѡ
						{
							Result1++;
						}
						else
							break;
					}	

					if(Result1==BandNum )
					{	

						T1=fabs(BufferIn55[b][r1+Width*s1]-BufferIn44[b][r1+Width*s1]);
						S1=fabs(BufferIn33[b][r1+Width*s1]-BufferIn44[b][r1+Width*s1]);
						if(S1<fabs(BufferIn33[b][i+Width*j]-BufferIn44[b][i+Width*j])+sqrt(M_err*M_err+L_err*L_err)||fabs(T1-fabs(BufferIn55[b][i+Width*j]-BufferIn44[b][i+Width*j]))<M_err*sqrt(2.0))
						{
							location_p[n1+n2+Idx*Win_size1*Win_size1]=r1+Width*s1;
							n2++;
						}
					}
				}
			}
			if(n1>5&&n2>5)
			{
				sumx=0;
				sumy=0;
				sumxy=0;
				sumx2=0;
				for(k_h=0;k_h<n1;k_h++)
				{
					sumxy+=BufferIn22[b][location_p[k_h+Idx*Win_size1*Win_size1]]*BufferIn55[b][location_p[k_h+Idx*Win_size1*Win_size1]];
					sumx+=BufferIn22[b][location_p[k_h+Idx*Win_size1*Win_size1]];
					sumy+=BufferIn55[b][location_p[k_h+Idx*Win_size1*Win_size1]];
					sumx2+=BufferIn22[b][location_p[k_h+Idx*Win_size1*Win_size1]]*BufferIn22[b][location_p[k_h+Idx*Win_size1*Win_size1]];
				}
				sumxy+=gamma;
				sumx2+=gamma;
				aa=(n1*sumxy-sumx*sumy)/(n1*sumx2-sumx*sumx);
				bb=sumy/n1-aa*sumx/n1; 
				sumx=0;
				sumy=0;
				sumxy=0;
				sumx2=0;
				for(k_h=0;k_h<n2;k_h++)
				{
					sumxy+=BufferIn44[b][location_p[n1+k_h+Idx*Win_size1*Win_size1]]*BufferIn55[b][location_p[n1+k_h+Idx*Win_size1*Win_size1]];
					sumx+=BufferIn44[b][location_p[n1+k_h+Idx*Win_size1*Win_size1]];
					sumy+=BufferIn55[b][location_p[n1+k_h+Idx*Win_size1*Win_size1]];
					sumx2+=BufferIn44[b][location_p[n1+k_h+Idx*Win_size1*Win_size1]]*BufferIn44[b][location_p[n1+k_h+Idx*Win_size1*Win_size1]];
				}
				sumxy+=gamma;
				sumx2+=gamma;
				aa2=(n2*sumxy-sumx*sumy)/(n2*sumx2-sumx*sumx);
				bb2=sumy/n2-aa2*sumx/n2; 
			}
			else
			{
				sumx=0;
				sumy=0;
				sumxy=0;
				sumx2=0;
				for(r1=rmin;r1<=rmax;r1++)
				{
					for( s1=smin;s1<=smax;s1++)
					{
						sumxy+=BufferIn22[b][r1+Width*s1]*BufferIn55[b][r1+Width*s1];
						sumx+=BufferIn22[b][r1+Width*s1];
						sumy+=BufferIn55[b][r1+Width*s1];
						sumx2+=BufferIn22[b][r1+Width*s1]*BufferIn22[b][r1+Width*s1];
					}
				}
				sumxy+=gamma;
				sumx2+=gamma;
				aa=((rmax-rmin+1)*(smax-smin+1)*sumxy-sumx*sumy)/((rmax-rmin+1)*(smax-smin+1)*sumx2-sumx*sumx);
				bb=sumy/((rmax-rmin+1)*(smax-smin+1))-aa*sumx/((rmax-rmin+1)*(smax-smin+1)); 
				sumx=0;
				sumy=0;
				sumxy=0;
				sumx2=0;
				for(r1=rmin;r1<=rmax;r1++)
				{
					for( s1=smin;s1<=smax;s1++)
					{
						sumxy+=BufferIn44[b][r1+Width*s1]*BufferIn55[b][r1+Width*s1];
						sumx+=BufferIn44[b][r1+Width*s1];
						sumy+=BufferIn55[b][r1+Width*s1];
						sumx2+=BufferIn44[b][r1+Width*s1]*BufferIn44[b][r1+Width*s1];
					}
				}
				sumxy+=gamma;
				sumx2+=gamma;
				aa2=((rmax-rmin+1)*(smax-smin+1)*sumxy-sumx*sumy)/((rmax-rmin+1)*(smax-smin+1)*sumx2-sumx*sumx);
				bb2=sumy/((rmax-rmin+1)*(smax-smin+1))-aa2*sumx/((rmax-rmin+1)*(smax-smin+1)) ; 
			}
			Changed_BufferIn11[kkk]=BufferIn11[b][j*Width+i]*aa+bb;
			Changed_BufferIn33[kkk]=BufferIn33[b][j*Width+i]*aa2+bb2;
			}
		}
	}
}
__global__ void No_limit_a_CalcuRela(float **BufferIn11,float **BufferIn22,float **BufferIn33,float **BufferIn44,float **BufferIn55, int Height,int Width,int BandNum,float *Changed_BufferIn11,float *Changed_BufferIn33)
{
	const int tid=threadIdx.x;
	const int bid=blockIdx.x;
	const int Idx=num_thread*bid+tid;
	int b=Idx/(Height*Width);
	int j=(Idx-(Height*Width)*b)/Width;
	int i=(Idx-(Height*Width)*b)%Width;
	for(int kkk=Idx;kkk<Width*Height*BandNum;kkk=kkk+num_thread*num_block)
	{
		b=kkk/(Height*Width);
		j=(kkk-(Height*Width)*b)/Width;
		i=(kkk-(Height*Width)*b)%Width;
		Changed_BufferIn11[kkk]=BufferIn11[b][j*Width+i]+BufferIn55[b][j*Width+i]-BufferIn22[b][j*Width+i];
		Changed_BufferIn33[kkk]=BufferIn33[b][j*Width+i]+BufferIn55[b][j*Width+i]-BufferIn44[b][j*Width+i];
	}
}
__global__ void No_limit_a_CalcuRela2(float **BufferIn11,float **BufferIn22,float **BufferIn55, int Height,int Width,int BandNum,float *Changed_BufferIn11)
{
	const int tid=threadIdx.x;
	const int bid=blockIdx.x;
	const int Idx=num_thread*bid+tid;
	int b=Idx/(Height*Width);
	int j=(Idx-(Height*Width)*b)/Width;
	int i=(Idx-(Height*Width)*b)%Width;
	for(int kkk=Idx;kkk<Width*Height*BandNum;kkk=kkk+num_thread*num_block)
	{
		b=kkk/(Height*Width);
		j=(kkk-(Height*Width)*b)/Width;
		i=(kkk-(Height*Width)*b)%Width;
		Changed_BufferIn11[kkk]=BufferIn11[b][j*Width+i]+BufferIn55[b][j*Width+i]-BufferIn22[b][j*Width+i];
	}
}
__global__ void limit_a_CalcuRela2(float **BufferIn11,float **BufferIn22,float **BufferIn55, int Height,int Width, int Win_size1,int flag,double L_err,double M_err,double h1,int BandNum,double gamma,int *location_p,float *Changed_BufferIn11,int p_Para,float d_Para)
{
	const int tid=threadIdx.x;
	const int bid=blockIdx.x;
	const int Idx=num_thread*bid+tid;
	int Result1=0,m=0;
	int b=Idx/(Height*Width);
	int j=(Idx-(Height*Width)*b)/Width;
	int i=(Idx-(Height*Width)*b)%Width;
	int rmin,rmax,smin,smax;
	int n1=0;
	float T1=0,S1=0;
   float threshold_d[10];
	//float B1[2]={1,0};
	//float B2[2]={gamma,0};
	//float temp2[4],temp22[4],temp33[4],tempB1[2],tempB11[2];
	//float temp3[2],temp11[2];
	int k_h=0;
	//float output[4];
	int r1,s1;
		float sumx,sumy,sumxy,sumx2;
	float aa,bb;
	for(int kkk=Idx;kkk<Width*Height*BandNum;kkk=kkk+num_thread*num_block)
	{
	//	B1[0]=1,B1[1]=0;
	//	B2[0]=1,B2[1]=0;
		aa=0,bb=0;
		Result1=0,m=0;
		b=kkk/(Height*Width);
		j=(kkk-(Height*Width)*b)/Width;
		i=(kkk-(Height*Width)*b)%Width;
		n1=0;
		T1=0,S1=0;
		k_h=0;
			sumx=0;
			sumy=0;
		     sumxy=0;
			sumx2=0;
		if(b<BandNum)
		{
			if(fabs(BufferIn11[b][j*Width+i])<1e-6)
			{
				Changed_BufferIn11[kkk]=0;
			}
			else
			{
			if(i-Win_size1/2<=0)
				rmin=0;
			else
				rmin = i-Win_size1/2;

			if(i+Win_size1/2>=Width-1)
				rmax = Width-1;
			else
				rmax = i+Win_size1/2;

			if(j-Win_size1/2<=0)
				smin=0;
			else
				smin = j-Win_size1/2;

			if(j+Win_size1/2>=Height-1)
				smax = Height-1;
			else
				smax = j+Win_size1/2;
			//Location_pp(location_p,BufferIn11,BufferIn22,BufferIn55,i, j, rmin, rmax, smin, smax, Width,Height,Win_size1, b, BandNum,n1);
			r1=rmin,s1=smin;
			for ( m=0;m<BandNum;m++)
			{
				if(p_Para==1||p_Para==2)
				{
				threshold_d[m]=d_Para*pow((float)2,BufferIn11[m][ i+Width*j]);
				}
				else if(p_Para==3)
				{
					threshold_d[m]=d_Para*pow((float)2,BufferIn11[m][ i+Width*j]/1000);
				}
			}

			for(r1=rmin;r1<=rmax;r1++)
			{
				for(s1=smin;s1<=smax;s1++)
				{  

					Result1=0;	
					for( m=0;m<BandNum;m++)
					{  
						if(fabs(BufferIn11[m][ r1+Width*s1]-BufferIn11[m][ i+Width*j])<=threshold_d[m])//ɸѡ
						{
							Result1++;
						}
						else
							break;
					}	

					if(Result1==BandNum )
					{	

						T1=fabs(BufferIn55[b][r1+Width*s1]-BufferIn22[b][r1+Width*s1]);
						S1=fabs(BufferIn11[b][r1+Width*s1]-BufferIn22[b][r1+Width*s1]);
						if(S1<fabs(BufferIn11[b][i+Width*j]-BufferIn22[b][i+Width*j])+sqrt(M_err*M_err+L_err*L_err)||fabs(T1-fabs(BufferIn55[b][i+Width*j]-BufferIn22[b][i+Width*j]))<M_err*sqrt(2.0))

						{
							location_p[n1+Idx*Win_size1*Win_size1]=r1+Width*s1;
							n1++;
						}
					}
				}
			}
			if(n1>5)
			{
				sumx=0;
				sumy=0;
				sumxy=0;
				sumx2=0;
				for(k_h=0;k_h<n1;k_h++)
				{
					sumxy+=BufferIn22[b][location_p[k_h+Idx*Win_size1*Win_size1]]*BufferIn55[b][location_p[k_h+Idx*Win_size1*Win_size1]];
					sumx+=BufferIn22[b][location_p[k_h+Idx*Win_size1*Win_size1]];
					sumy+=BufferIn55[b][location_p[k_h+Idx*Win_size1*Win_size1]];
					sumx2+=BufferIn22[b][location_p[k_h+Idx*Win_size1*Win_size1]]*BufferIn22[b][location_p[k_h+Idx*Win_size1*Win_size1]];
				}
				sumxy+=gamma;
				sumx2+=gamma;
				aa=(n1*sumxy-sumx*sumy)/(n1*sumx2-sumx*sumx);
				bb=sumy/n1-aa*sumx/n1;
			}
			else
			{
				sumx=0;
				sumy=0;
				sumxy=0;
				sumx2=0;
				for(r1=rmin;r1<=rmax;r1++)
				{
					for( s1=smin;s1<=smax;s1++)
					{
						sumxy+=BufferIn22[b][r1+Width*s1]*BufferIn55[b][r1+Width*s1];
						sumx+=BufferIn22[b][r1+Width*s1];
						sumy+=BufferIn55[b][r1+Width*s1];
						sumx2+=BufferIn22[b][r1+Width*s1]*BufferIn22[b][r1+Width*s1];
					}
				}
				sumxy+=gamma;
				sumx2+=gamma;
				aa=((rmax-rmin+1)*(smax-smin+1)*sumxy-sumx*sumy)/((rmax-rmin+1)*(smax-smin+1)*sumx2-sumx*sumx);
				bb=sumy/((rmax-rmin+1)*(smax-smin+1))-aa*sumx/((rmax-rmin+1)*(smax-smin+1)); 
			}
			Changed_BufferIn11[kkk]=BufferIn11[b][j*Width+i]*aa+bb;
			}
		}
	}
}
 int runtest1(float **BufferIn11,float **BufferIn22,float **BufferIn33,float **BufferIn44,float **BufferIn55, float **BufferOut,int Height,int Width, int Win_size1,int flag,double L_err,double M_err,double h1,int BandNum,double gamma,int p_Para,float d_Para,int patchSize)
{
	float **dev_BufferIn11,**dev_BufferIn22,**dev_BufferIn33,**dev_BufferIn44,**dev_BufferIn55,**dev_BufferOut,*Changed_BufferIn11,*Changed_BufferIn33;
	float **a,**f,**c,**d,**e,**out;
	//float *AA,*BB;
	a = (float**)malloc(BandNum*sizeof(float*));
	f = (float**)malloc(BandNum*sizeof(float*));
	c = (float**)malloc(BandNum*sizeof(float*));
	d = (float**)malloc(BandNum*sizeof(float*));
	e = (float**)malloc(BandNum*sizeof(float*));
	out=(float**)malloc(BandNum*sizeof(float*));
	for(int b=0;b<BandNum;b++)
	{
		hipMalloc((void**)&a[b],Height*Width*sizeof(float));
		hipMalloc((void**)&f[b],Height*Width*sizeof(float));
		hipMalloc((void**)&c[b],Height*Width*sizeof(float));
		hipMalloc((void**)&d[b],Height*Width*sizeof(float));
		hipMalloc((void**)&e[b],Height*Width*sizeof(float));
		hipMalloc((void**)&out[b],Height*Width*sizeof(float));

	}
	//int num_block= Height* Width*BandNum/num_thread+1;
	int *Location_P;
	float *GausKernel;
	hipMalloc((void***)&dev_BufferIn11,sizeof(float*)*BandNum);
	hipMalloc((void***)&dev_BufferIn22,sizeof(float*)*BandNum);
	hipMalloc((void***)&dev_BufferIn33,sizeof(float*)*BandNum);
	hipMalloc((void***)&dev_BufferIn44,sizeof(float*)*BandNum);
	hipMalloc((void***)&dev_BufferIn55,sizeof(float*)*BandNum);
	hipMalloc((void***)&dev_BufferOut,sizeof(float*)*BandNum);
	hipMalloc((void**)&Location_P,sizeof(float)*Win_size1*Win_size1*num_block*num_thread);
	hipMalloc((void**)&GausKernel,sizeof(float)*Win_size1*Win_size1*num_block*num_thread);
	//hipMalloc((void**)&AA,sizeof(float)*2*Win_size1*Win_size1*num_block*num_thread);
	//hipMalloc((void**)&temp1,sizeof(float)*2*Win_size1*Win_size1*num_block*num_thread);
//	hipMalloc((void**)&BB,sizeof(float)*Win_size1*Win_size1*num_block*num_thread);
	hipMalloc((void**)&Changed_BufferIn11,sizeof(float)*BandNum*Height*Width);
	hipMalloc((void**)&Changed_BufferIn33,sizeof(float)*BandNum*Height*Width);
	hipMemset(GausKernel,0,sizeof(float)*Win_size1*Win_size1*num_block*num_thread);
	//hipMalloc((void**)&Location_P,sizeof(float)*100*Width*Height*BandNum);
	for(int g=0;g<BandNum;g++)
	{
		hipMemcpy(a[g], BufferIn11[g],Height*Width*sizeof(float),hipMemcpyHostToDevice);
		hipMemcpy(f[g], BufferIn22[g],Height*Width*sizeof(float),hipMemcpyHostToDevice);
		hipMemcpy(c[g], BufferIn33[g],Height*Width*sizeof(float),hipMemcpyHostToDevice);
		hipMemcpy(d[g], BufferIn44[g],Height*Width*sizeof(float),hipMemcpyHostToDevice);
		hipMemcpy(e[g], BufferIn55[g],Height*Width*sizeof(float),hipMemcpyHostToDevice);
	}
	hipMemcpy(dev_BufferIn11,a,sizeof(float*)*BandNum,hipMemcpyHostToDevice);
	hipMemcpy(dev_BufferIn22,f,sizeof(float*)*BandNum,hipMemcpyHostToDevice);
	hipMemcpy(dev_BufferIn33, c,sizeof(float*)*BandNum,hipMemcpyHostToDevice);
	hipMemcpy(dev_BufferIn44, d,sizeof(float*)*BandNum,hipMemcpyHostToDevice);
	hipMemcpy(dev_BufferIn55, e,sizeof(float*)*BandNum,hipMemcpyHostToDevice);
	hipMemcpy(dev_BufferOut,out,sizeof(float*)*BandNum,hipMemcpyHostToDevice);
	if(flag==0)
	{
		No_limit_a_CalcuRela<<<num_block, num_thread>>>(dev_BufferIn11,dev_BufferIn22,dev_BufferIn33,dev_BufferIn44,dev_BufferIn55, Height, Width,BandNum,Changed_BufferIn11,Changed_BufferIn33);
	}
	else
	{
		limit_a_CalcuRela<<<num_block, num_thread>>>(dev_BufferIn11,dev_BufferIn22,dev_BufferIn33,dev_BufferIn44,dev_BufferIn55, Height, Width,  Win_size1, flag,L_err,M_err,h1,BandNum,gamma,Location_P,Changed_BufferIn11,Changed_BufferIn33,p_Para,d_Para);
	}
	blending2<<<num_block, num_thread>>>(dev_BufferIn11,dev_BufferIn22,dev_BufferIn33,dev_BufferIn44,dev_BufferIn55,dev_BufferOut, Height, Width,  Win_size1, flag,L_err,M_err,h1,BandNum,gamma,Location_P,Changed_BufferIn11,Changed_BufferIn33,GausKernel,p_Para,d_Para,patchSize);
	for(int g=0;g<BandNum;g++)
	{
		hipMemcpy(BufferOut[g],out[g],Height*Width*sizeof(float),hipMemcpyDeviceToHost);
	}
	for(int g=0;g<BandNum;g++)
	{
		hipFree(a[g]);
		hipFree(f[g]);
		hipFree(c[g]);
		hipFree(d[g]);
		hipFree(e[g]);
		hipFree(out[g]);
	}
	hipFree(Changed_BufferIn11);
	hipFree(Changed_BufferIn33);
	hipFree(Location_P);
//	hipFree(AA);
	//hipFree(BB);
	//hipFree(temp1);
	return 0;
}
  int runtest2(float **BufferIn11,float **BufferIn22,float **BufferIn55, float **BufferOut,int Height,int Width, int Win_size1,int flag,double L_err,double M_err,double h1,int BandNum,double gamma,int p_Para,float d_Para,int patchSize)
{
	float **dev_BufferIn11,**dev_BufferIn22,**dev_BufferIn55,**dev_BufferOut,*Changed_BufferIn11;
	float **a,**d,**e,**out;
	//float *AA,*BB;
	a = (float**)malloc(BandNum*sizeof(float*));
	d = (float**)malloc(BandNum*sizeof(float*));
	e = (float**)malloc(BandNum*sizeof(float*));
	out=(float**)malloc(BandNum*sizeof(float*));
	for(int b=0;b<BandNum;b++)
	{
		hipMalloc((void**)&a[b],Height*Width*sizeof(float));
		hipMalloc((void**)&d[b],Height*Width*sizeof(float));
		hipMalloc((void**)&e[b],Height*Width*sizeof(float));
		hipMalloc((void**)&out[b],Height*Width*sizeof(float));

	}
	//int num_block= Height* Width*BandNum/num_thread+1;
	int *Location_P;
	float *GausKernel;

	hipMalloc((void**)&GausKernel,sizeof(float)*Win_size1*Win_size1*num_block*num_thread);
	hipMalloc((void***)&dev_BufferIn11,sizeof(float*)*BandNum);
	hipMalloc((void***)&dev_BufferIn22,sizeof(float*)*BandNum);
	hipMalloc((void***)&dev_BufferIn55,sizeof(float*)*BandNum);
	hipMalloc((void***)&dev_BufferOut,sizeof(float*)*BandNum);
	hipMalloc((void**)&Location_P,sizeof(float)*Win_size1*Win_size1*num_block*num_thread);
	//hipMalloc((void**)&AA,sizeof(float)*2*Win_size1*Win_size1*num_block*num_thread);
	//hipMalloc((void**)&temp1,sizeof(float)*2*Win_size1*Win_size1*num_block*num_thread);
//	hipMalloc((void**)&BB,sizeof(float)*Win_size1*Win_size1*num_block*num_thread);
	hipMalloc((void**)&Changed_BufferIn11,sizeof(float)*BandNum*Height*Width);
	//hipMalloc((void**)&Changed_BufferIn33,sizeof(float)*BandNum*Height*Width);
	//hipMalloc((void**)&Location_P,sizeof(float)*100*Width*Height*BandNum);
	hipMemset(GausKernel,0,sizeof(float)*Win_size1*Win_size1*num_block*num_thread);
	for(int g=0;g<BandNum;g++)
	{
		hipMemcpy(a[g], BufferIn11[g],Height*Width*sizeof(float),hipMemcpyHostToDevice);
		hipMemcpy(d[g], BufferIn22[g],Height*Width*sizeof(float),hipMemcpyHostToDevice);
		hipMemcpy(e[g], BufferIn55[g],Height*Width*sizeof(float),hipMemcpyHostToDevice);
	}
	hipMemcpy(dev_BufferIn11,a,sizeof(float*)*BandNum,hipMemcpyHostToDevice);
	hipMemcpy(dev_BufferIn22,d,sizeof(float*)*BandNum,hipMemcpyHostToDevice);
	hipMemcpy(dev_BufferIn55, e,sizeof(float*)*BandNum,hipMemcpyHostToDevice);
	hipMemcpy(dev_BufferOut,out,sizeof(float*)*BandNum,hipMemcpyHostToDevice);
	if(flag==0)
	{
		No_limit_a_CalcuRela2<<<num_block, num_thread>>>(dev_BufferIn11,dev_BufferIn22,dev_BufferIn55, Height, Width,BandNum,Changed_BufferIn11);
	}
	else
	{
	limit_a_CalcuRela2<<<num_block, num_thread>>>(dev_BufferIn11,dev_BufferIn22,dev_BufferIn55, Height, Width,  Win_size1, flag,L_err,M_err,h1,BandNum,gamma,Location_P,Changed_BufferIn11,p_Para,d_Para);
	}
	blending3<<<num_block, num_thread>>>(dev_BufferIn11,dev_BufferIn22,dev_BufferIn55,dev_BufferOut, Height, Width,  Win_size1, flag,L_err,M_err,h1,BandNum,gamma,Location_P,Changed_BufferIn11,GausKernel,p_Para,d_Para,patchSize);
	for(int g=0;g<BandNum;g++)
	{
		hipMemcpy(BufferOut[g],out[g],Height*Width*sizeof(float),hipMemcpyDeviceToHost);
	}
	for(int g=0;g<BandNum;g++)
	{
		hipFree(a[g]);
		hipFree(d[g]);
		hipFree(e[g]);
		hipFree(out[g]);
	}
	hipFree(Changed_BufferIn11);
	//hipFree(Changed_BufferIn33);
	hipFree(Location_P);
//	hipFree(AA);
	//hipFree(BB);
	//hipFree(temp1);
	return 0;
}
 void IDWInterpolation(float **bufferFusResult, int rowTar, int colTar, int band, int height, int width)
{
	int numValPix = 0, halfWin = 1;
	double distIndex, sumDistIndex, sumWeigPred;

	while (numValPix <3)
	{
		numValPix = 0;
		sumDistIndex = 0;
		sumWeigPred = 0;

		/* 根据待插值像元的位置信息和搜索半径，确定搜索窗口的行列范围 */
		int rowMin = rowTar-halfWin<0 ? 0 : rowTar-halfWin;
		int colMin = colTar-halfWin<0 ? 0 : colTar-halfWin;
		int rowMax = rowTar+halfWin>height-1 ? height-1: rowTar+halfWin;
		int colMax = colTar+halfWin>width-1 ? width-1: colTar+halfWin;
		
		/* 遍历窗口内所有像元，对有效像元进行计数。使用有效像元信息，通过IDW对目标像元进行插值。*/
		for (int rowValPix = rowMin; rowValPix <= rowMax; rowValPix++)
		{
			for (int colValPix = colMin; colValPix <= colMax; colValPix++)
			{
				if(!_isnan(bufferFusResult[band][rowValPix*width+colValPix]))
				{
					distIndex = 1.0/(double)((rowTar-rowValPix)*(rowTar-rowValPix)+(colTar-colValPix)*(colTar-colValPix));
					sumWeigPred += distIndex * bufferFusResult[band][rowValPix*width+colValPix];
					sumDistIndex += distIndex;
					numValPix ++;
				}
			}
		}
		
		halfWin++;    // 更改搜索窗口半径
	}
	bufferFusResult[band][rowTar*width+colTar] = sumWeigPred / sumDistIndex;

}
 void runtest_one(float **BufferIn11,float **BufferIn22,float **BufferIn55,float **BufferOut,int Height,int Width,int Win_size1,int flag,float L_err,float M_err,float Para_h,int BandNum,float gamma,int p_Para,float d_Para ,int patchSize)
{
	int maxnum;
	size_t ff,tt;
	//hipSetDevice(0);
	hipMemGetInfo(&ff, &tt);
	maxnum=(ff-sizeof(float)*Win_size1*Win_size1*num_block*num_thread*2)/(BandNum*sizeof(float)*5);
	int sub_height=maxnum/Width-Win_size1;
//	sub_height=100;
	int kk=0;
	int i,j;
	float **sub_BufferIn11,**sub_BufferIn22,**sub_BufferIn55,**sub_out;
	for(int heiht_all=0;heiht_all<Height;heiht_all+=sub_height)
	{
		int task_start=kk*sub_height;
		int task_end;
		if((kk+1)*sub_height-Height<=0)
			task_end=(kk+1)*sub_height-1;
		else
			task_end=Height-1; 
		int data_start,data_end;
		if(task_start-Win_size1+1<=0)
			data_start= 0;
		else
			data_start=task_start-Win_size1+1;
		if(task_end+Win_size1-1>=Height-1)
			data_end=Height-1;
		else
			data_end=task_end+Win_size1-1;
		int data_height=data_end-data_start+1;
		sub_BufferIn11=(float**)malloc(BandNum*sizeof(float*));
		sub_BufferIn22=(float**)malloc(BandNum*sizeof(float*));
		sub_BufferIn55=(float**)malloc(BandNum*sizeof(float*));
		sub_out=(float**)malloc(BandNum*sizeof(float*));
		for(int b=0;b<BandNum;b++)
		{
			sub_BufferIn11[b]=new float[data_height*Width];
			sub_BufferIn22[b]=new float[data_height*Width];
			sub_BufferIn55[b]=new float[data_height*Width];
			sub_out[b]=new float[data_height*Width];
		}
		int copy;
		for(int k=0;k<BandNum;k++)
		{
			copy=0;
			for( i=data_start;i<=data_end;i++)
			{
				for( j=0;j<Width;j++)
				{
					sub_BufferIn11[k][copy*Width+j]=BufferIn11[k][i*Width+j];
					sub_BufferIn22[k][copy*Width+j]=BufferIn22[k][i*Width+j];
					sub_BufferIn55[k][copy*Width+j]=BufferIn55[k][i*Width+j];
				}
				copy++;
			}
		}
		int current=task_start-data_start;
		runtest2(sub_BufferIn11,sub_BufferIn22,sub_BufferIn55,sub_out,data_height,Width,Win_size1,flag, L_err, M_err, Para_h,BandNum,1.0,p_Para,d_Para,patchSize);
		
		for(int k=0;k<BandNum;k++)
		{
			current=task_start-data_start;
			for(int i=task_start;i<=task_end;i++)
			{
				for(int j=0;j<Width;j++)
				{
					BufferOut[k][i*Width+j]=sub_out[k][current*Width+j];
				}
				current++;
			}
		}
		for(int g=0;g<BandNum;g++)
	{
		delete sub_BufferIn11[g];
		delete sub_BufferIn22[g];
		delete sub_BufferIn55[g];
		delete sub_out[g];
		/*hipFree(dev_BufferIn11[g]);
		hipFree(dev_BufferIn22[g]);
		hipFree(dev_BufferIn33[g]);
		hipFree(dev_BufferIn44[g]);
		hipFree(dev_BufferIn55[g]);
		hipFree(dev_BufferOut[g]);*/
	}
		kk++;
	}
	for(int b=0; b<BandNum; b++)
	{ 
		for(int j=0; j<Height; j++)
		{
			for(int i=0; i<Width;i++)
			{
				if (_isnan(BufferOut[b][j*Width+i]))
				{
					/* 对无效值位置处进行IDW插值 */
					IDWInterpolation(BufferOut, j, i, b, Height, Width);
				}
			}
		}
	}
}
 void runtest(float **BufferIn11,float **BufferIn22,float **BufferIn33,float **BufferIn44,float **BufferIn55,float **BufferOut,int Height,int Width,int Win_size1,int flag,float L_err,float M_err,float Para_h,int BandNum,float gamma,int p_Para,float d_Para ,int patchSize)
{
	int maxnum;
	size_t ff,tt;
	//hipSetDevice(0);
	hipMemGetInfo(&ff, &tt);
	maxnum=(ff-sizeof(float)*Win_size1*Win_size1*num_block*num_thread*2)/(BandNum*sizeof(float)*8);
	int sub_height=maxnum/Width-Win_size1;
//	sub_height=100;
	int kk=0;
	int i,j;
	float **sub_BufferIn11,**sub_BufferIn22,**sub_BufferIn33,**sub_BufferIn44,**sub_BufferIn55,**sub_out;
	for(int heiht_all=0;heiht_all<Height;heiht_all+=sub_height)
	{
		int task_start=kk*sub_height;
		int task_end;
		if((kk+1)*sub_height-Height<=0)
			task_end=(kk+1)*sub_height-1;
		else
			task_end=Height-1; 
		int data_start,data_end;
		if(task_start-Win_size1+1<=0)
			data_start= 0;
		else
			data_start=task_start-Win_size1+1;
		if(task_end+Win_size1-1>=Height-1)
			data_end=Height-1;
		else
			data_end=task_end+Win_size1-1;
		int data_height=data_end-data_start+1;
		sub_BufferIn11=(float**)malloc(BandNum*sizeof(float*));
		sub_BufferIn22=(float**)malloc(BandNum*sizeof(float*));
		sub_BufferIn33=(float**)malloc(BandNum*sizeof(float*));
		sub_BufferIn44=(float**)malloc(BandNum*sizeof(float*));
		sub_BufferIn55=(float**)malloc(BandNum*sizeof(float*));
		sub_out=(float**)malloc(BandNum*sizeof(float*));
		for(int b=0;b<BandNum;b++)
		{
			sub_BufferIn11[b]=new float[data_height*Width];
			sub_BufferIn22[b]=new float[data_height*Width];
			sub_BufferIn33[b]=new float[data_height*Width];
			sub_BufferIn44[b]=new float[data_height*Width];
			sub_BufferIn55[b]=new float[data_height*Width];
			sub_out[b]=new float[data_height*Width];
		}
		int copy;
		for(int k=0;k<BandNum;k++)
		{
			copy=0;
			for( i=data_start;i<=data_end;i++)
			{
				for( j=0;j<Width;j++)
				{
					sub_BufferIn11[k][copy*Width+j]=BufferIn11[k][i*Width+j];
					sub_BufferIn22[k][copy*Width+j]=BufferIn22[k][i*Width+j];
					sub_BufferIn33[k][copy*Width+j]=BufferIn33[k][i*Width+j];
					sub_BufferIn44[k][copy*Width+j]=BufferIn44[k][i*Width+j];
					sub_BufferIn55[k][copy*Width+j]=BufferIn55[k][i*Width+j];
				}
				copy++;
			}
		}
		int current=task_start-data_start;
		runtest1(sub_BufferIn11,sub_BufferIn22,sub_BufferIn33,sub_BufferIn44,sub_BufferIn55,sub_out,data_height,Width,Win_size1,flag, L_err, M_err, Para_h,BandNum,1.0,p_Para,d_Para,patchSize);
		
		for(int k=0;k<BandNum;k++)
		{
			current=task_start-data_start;
			for(int i=task_start;i<=task_end;i++)
			{
				for(int j=0;j<Width;j++)
				{
					BufferOut[k][i*Width+j]=sub_out[k][current*Width+j];
				}
				current++;
			}
		}
		for(int g=0;g<BandNum;g++)
	{
		delete sub_BufferIn11[g];
		delete sub_BufferIn22[g];
		delete sub_BufferIn33[g];
		delete sub_BufferIn44[g];
		delete sub_BufferIn55[g];
		delete sub_out[g];
		/*hipFree(dev_BufferIn11[g]);
		hipFree(dev_BufferIn22[g]);
		hipFree(dev_BufferIn33[g]);
		hipFree(dev_BufferIn44[g]);
		hipFree(dev_BufferIn55[g]);
		hipFree(dev_BufferOut[g]);*/
	}
		kk++;
	}
	for(int b=0; b<BandNum; b++)
	{ 
		for(int j=0; j<Height; j++)
		{
			for(int i=0; i<Width;i++)
			{
				if (_isnan(BufferOut[b][j*Width+i]))
				{
					/* 对无效值位置处进行IDW插值 */
					IDWInterpolation(BufferOut, j, i, b, Height, Width);
				}
			}
		}
	}
}
 void Re_fusion(CuLayer *psensor,PARAMETER *par)
 {
	 int c;
	long now1 = clock();
	for(c=0;c<par->NUM_PREDICTIONS;c++)
	{
		psensor[2*(par->NUM_PAIRS+c)].Read(psensor[2*(par->NUM_PAIRS+c)].outpath);
		psensor[2*(par->NUM_PAIRS+c)+1].resize(psensor[0].getWidth(),psensor[0].getHeight(),psensor[0].getbandCount());
		if(par->NUM_PAIRS==2)
		{
			runtest(psensor[0].getData(),psensor[2].getData(),psensor[1].getData(),psensor[3].getData(),psensor[2*(par->NUM_PAIRS+c)].getData(),psensor[2*(par->NUM_PAIRS+c)+1].getData(),psensor[0].getHeight(),psensor[0].getWidth(),par->WIN_SIZE,par->r,par->L_ERR,par->M_ERR,par->h,psensor[0].getbandCount(),par->gamma,par->p,par->d,par->pathSize);
			//char* driverName = "GTiff";
			psensor[2*(par->NUM_PAIRS+c)+1].setGeoTransform(psensor[0].getGeoTransform());
			psensor[2*(par->NUM_PAIRS+c)+1].setProjection(psensor[0].getProjection());
			psensor[2*(par->NUM_PAIRS+c)+1].Write(psensor[2*(par->NUM_PAIRS+c)+1].outpath,par->G_Type);
		}
		else if(par->NUM_PAIRS==1)
		{
			runtest_one(psensor[0].getData(),psensor[1].getData(),psensor[2*(par->NUM_PAIRS+c)].getData(),psensor[2*(par->NUM_PAIRS+c)+1].getData(),psensor[0].getHeight(),psensor[0].getWidth(),par->WIN_SIZE,par->r,par->L_ERR,par->M_ERR,par->h,psensor[0].getbandCount(),par->gamma,par->p,par->d,par->pathSize);
			psensor[2*(par->NUM_PAIRS+c)+1].setGeoTransform(psensor[0].getGeoTransform());
			psensor[2*(par->NUM_PAIRS+c)+1].setProjection(psensor[0].getProjection());
			psensor[2*(par->NUM_PAIRS+c)+1].Write(psensor[2*(par->NUM_PAIRS+c)+1].outpath,par->G_Type);
		}
		psensor[2*(par->NUM_PAIRS+c)].ddata();
		psensor[2*(par->NUM_PAIRS+c)+1].ddata();
		//delete &psensor[2*(par->NUM_PAIRS+c)];
		//delete &psensor[2*(par->NUM_PAIRS+c)+1];
	}
	 printf("GPU运行时间为：%dms\n", int(((double)(clock() - now1)) / CLOCKS_PER_SEC * 1000));
 }
//void Re_fusion2(const char * BufferIn0,const char * BufferIn1,const char * BufferIn2,const char * BufferIn3,const char * BufferIn4,const char * BufferOut,int win_size,int flag,float L_err,float M_err,float Para_h)
//{
//	GDALAllRegister();
//	CPLSetConfigOption("GDAL_FILENAME_IS_UTF8","NO"); 
//	GDALDataset *Landsat0 = (GDALDataset*) GDALOpen(BufferIn0,GA_ReadOnly);
//	int width,height,BandNum;
//	width = Landsat0->GetRasterXSize();
//	height = Landsat0->GetRasterYSize();
//	BandNum = Landsat0->GetRasterCount();
//	float** BufferLandsat_0 = new float*[BandNum];
//	int b,k;
//	for( b=0;b<BandNum;b++)
//	{
//		BufferLandsat_0[b] = new float[width*height];	
//	}
//	
//	for( k=0;k<BandNum;k++)
//	{
//		GDALRasterBand* hInBand1 = Landsat0->GetRasterBand(k+1);
//		hInBand1->RasterIO(GF_Read,0,0,width,height,BufferLandsat_0[k],width,height,GDT_Float32,0,0);
//	}	
//
//	GDALDataset *MODIS0 = (GDALDataset*) GDALOpen(BufferIn1,GA_ReadOnly);
//	float** BufferModis_0 = new float*[BandNum];
//	for( b=0;b<BandNum;b++)
//	{
//		BufferModis_0[b] = new float[width*height];	
//	}
//	
//	for( k=0;k<BandNum;k++)
//	{
//		GDALRasterBand* hInBand1 = MODIS0->GetRasterBand(k+1);
//		hInBand1->RasterIO(GF_Read,0,0,width,height,BufferModis_0[k],width,height,GDT_Float32,0,0);		
//	}	
//
//	GDALDataset *Landsat1 = (GDALDataset*) GDALOpen(BufferIn2,GA_ReadOnly);
//	float** BufferLandsat_1 = new float*[BandNum];
//	for( b=0;b<BandNum;b++)
//	{
//		BufferLandsat_1[b] = new float[width*height];	
//	}
//	
//	for(k=0;k<BandNum;k++)
//	{
//		GDALRasterBand* hInBand1 = Landsat1->GetRasterBand(k+1);
//		hInBand1->RasterIO(GF_Read,0,0,width,height,BufferLandsat_1[k],width,height,GDT_Float32,0,0);
//	}	
//
//	GDALDataset *MODIS1 = (GDALDataset*) GDALOpen(BufferIn3,GA_ReadOnly);
//	float** BufferModis_1 = new float*[BandNum];
//	for( b=0;b<BandNum;b++)
//	{
//		BufferModis_1[b] = new float[width*height];	
//	}
//	
//	for( k=0;k<BandNum;k++)
//	{
//		GDALRasterBand* hInBand1 = MODIS1->GetRasterBand(k+1);
//		hInBand1->RasterIO(GF_Read,0,0,width,height,BufferModis_1[k],width,height,GDT_Float32,0,0);		
//	}	
//	
//	GDALDataset *MODIS2 = (GDALDataset*) GDALOpen(BufferIn4,GA_ReadOnly);
//	
//	float** BufferModis_2 = new float*[BandNum];
//	for( b=0;b<BandNum;b++)
//	{
//		BufferModis_2[b] = new float[width*height];	
//	}
//	
//	for( k=0;k<BandNum;k++)
//	{
//		GDALRasterBand* hInBand1 = MODIS2->GetRasterBand(k+1);
//		hInBand1->RasterIO(GF_Read,0,0,width,height,BufferModis_2[k],width,height,GDT_Float32,0,0);		
//		
//	}
//	
//	GDALDataset *LandsatDs;
//	char* driverName = "GTiff";
//	GDALDriver *pDriver = (GDALDriver*)GDALGetDriverByName(driverName);
//	LandsatDs = pDriver->Create(BufferOut,width,height,BandNum,GDT_Float64,NULL);
//	double* geos=new double[6];
//	Landsat0->GetGeoTransform(geos);
//	LandsatDs->SetGeoTransform(geos);
//	LandsatDs->SetProjection(Landsat0->GetProjectionRef());
//	
//	float** BufferOutColor = new float*[BandNum];
//	for( b=0;b<BandNum;b++)
//	{
//		BufferOutColor[b] = new float[width*height];
//	}
//	//e.Blending2(BufferLandsat_0,BufferModis_0,BufferLandsat_1,BufferModis_1,BufferModis_2,BufferOutColor,height,width,win_size,flag, L_err, M_err, Para_h,BandNum,1.0);
//	long now1 = clock();
//	 runtest(BufferLandsat_0,BufferModis_0,BufferLandsat_1,BufferModis_1,BufferModis_2,BufferOutColor,height,width,win_size,flag, L_err, M_err, Para_h,BandNum,1.0);
//	 printf("GPU运行时间为：%dms\n", int(((double)(clock() - now1)) / CLOCKS_PER_SEC * 1000));
//	for (b=0;b<BandNum;b++)
//	{
//		GDALRasterBand* HOut = LandsatDs->GetRasterBand(b+1);
//		HOut->RasterIO(GF_Write,0,0,width,height,BufferOutColor[b],width,height,GDT_Float32,0,0);
//	}
//	GDALClose(Landsat0);
//	GDALClose(MODIS0);
//	GDALClose(Landsat1);
//	GDALClose(MODIS1);
//	GDALClose(MODIS2);
//	GDALClose(LandsatDs);
//
//	for (b=0;b<BandNum;b++)
//	{
//		delete []BufferLandsat_0[b];
//		delete []BufferModis_0[b];
//		delete []BufferLandsat_1[b];
//		delete []BufferModis_1[b];
//		delete []BufferModis_2[b];
//		delete []BufferOutColor[b];
//	}
//	delete []BufferLandsat_0;
//	delete [] BufferModis_0;
//	delete []BufferLandsat_1;
//	delete [] BufferModis_1;
//	delete [] BufferModis_2;
//	delete [] BufferOutColor;
//}
//int main()
//{
//	const char* modFile1="D:\\cuda\\shikong\\软件\\测试数据\\M_2002_01_04.tif";
//    const char* modFile2="D:\\cuda\\shikong\\软件\\测试数据\\M_2002_02_21.tif";
//	const char* tifFile1="D:\\cuda\\shikong\\软件\\测试数据\\L_2002_01_04.tif";
//	const char* tifFile2="D:\\cuda\\shikong\\软件\\测试数据\\L_2002_02_21.tif";
//
//	const char* modFile0="D:\\cuda\\shikong\\软件\\测试数据\\M_2002_02_12.tif";
//	const char* out="D:\\cuda\\shikong\\软件\\测试数据\\kk_Llick_2001_11_02.tif";
//	Re_fusion2(tifFile1, modFile1,tifFile2,modFile2,modFile0,out,51,1,0.0028,0.0028,0.03);
//	hipDeviceReset();
//	return 0;
//}